#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include "ambocc.h"

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal,   attribute shading_normal, ); 

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow,   rtPayload, );

rtDeclareVariable(optix::Ray, ray,          rtCurrentRay, );
rtDeclareVariable(float,      t_hit,        rtIntersectionDistance, );
rtDeclareVariable(uint2,      launch_index, rtLaunchIndex, );

rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(unsigned int, shadow_ray_type , , );
rtDeclareVariable(float,        scene_epsilon, , );
rtDeclareVariable(rtObject,     top_object, , );




// Create ONB from normal.  Resulting W is Parallel to normal
__device__ __inline__ void createONB( const optix::float3& n,
    optix::float3& U,
    optix::float3& V,
    optix::float3& W )
{
  using namespace optix;

  W = normalize( n );
  U = cross( W, make_float3( 0.0f, 1.0f, 0.0f ) );
  if ( fabsf( U.x) < 0.001f && fabsf( U.y ) < 0.001f && fabsf( U.z ) < 0.001f  )
    U = cross( W, make_float3( 1.0f, 0.0f, 0.0f ) );
  U = normalize( U );
  V = cross( W, U );
}

// Create ONB from normalalized vector
__device__ __inline__ void createONB( const optix::float3& n,
    optix::float3& U,
    optix::float3& V)
{
  using namespace optix;
  U = cross( n, make_float3( 0.0f, 1.0f, 0.0f ) );
  if ( dot(U, U) < 1.e-3f )
    U = cross( n, make_float3( 1.0f, 0.0f, 0.0f ) );
  U = normalize( U );
  V = cross( n, U );
}

// sample hemisphere with cosine density
__device__ __inline__ void sampleUnitHemisphere( const optix::float2& sample,
    const optix::float3& U,
    const optix::float3& V,
    const optix::float3& W,
    optix::float3& point )
{
  using namespace optix;

  float phi = 2.0f * M_PIf*sample.x;
  float r = sqrt( sample.y );
  float x = r * cos(phi);
  float y = r * sin(phi);
  float z = 1.0f - x*x -y*y;
  z = z > 0.0f ? sqrt(z) : 0.0f;

  point = x*U + y*V + z*W;
}

rtBuffer<float, 1>              gaussian_lookup;

__device__ __inline__ float gaussFilter(float dist, float zmin)
{

  //float scale = 0.5;                  //scale = 2*z_min*omegaShadeMax /omegaVMax
  float scale = zmin/2.0;
  float sample = dist/scale;
  if (sample > 0.9999) {
    return 0.0;
  }
  float scaled = sample*64;
  int index = (int) scaled;
  float weight = scaled - index;
  return (1.0 - weight) * gaussian_lookup[index] + weight * gaussian_lookup[index + 1]; 
}

//
// Pinhole camera implementation
//
rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtBuffer<uchar4, 2>              output_buffer;

rtDeclareVariable(float3, bg_color, , );

rtBuffer<float4, 2>              accum_buffer;
rtBuffer<float4, 2>              accum_buffer_occ;
//rtBuffer<float4, 2>              accum_buffer_occ_h;
rtBuffer<float3, 2>              world_loc;
rtBuffer<float, 2>               closest_intersection;
rtDeclareVariable(uint,           frame, , );
rtDeclareVariable(uint,           blur_occ, , );
rtDeclareVariable(uint,           err_vis, , );

rtDeclareVariable(uint,           normal_rpp, , );
rtDeclareVariable(uint,           brute_rpp, , );
rtDeclareVariable(uint,           show_progressive, , );
rtDeclareVariable(int2,          pixel_radius, , );

RT_PROGRAM void pinhole_camera() {

  size_t2 screen = output_buffer.size();

  float2 d = make_float2(launch_index) / make_float2(screen) * 2.f - 1.f;
  float3 ray_origin = eye;
  float3 ray_direction = normalize(d.x*U + d.y*V + W);

  if(frame == 0)
    closest_intersection[launch_index] = 10;


  float zmin = closest_intersection[launch_index];
  if (zmin < 0) {
    output_buffer[launch_index] = make_color(bg_color);
    return;
  }

  bool newInfo = false;
  if (frame < normal_rpp)
    newInfo = true;
  else if (zmin < 0.05 && frame < brute_rpp)
    newInfo = true;


  float4 acc_val = accum_buffer[launch_index];
  float4 acc_val_occ = accum_buffer_occ[launch_index];
  float3 cur_world_loc = make_float3(0.0);

  if (newInfo) {

    optix::Ray ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon );

    PerRayData_radiance prd;
    prd.importance = 1.f;
    prd.occlusion = make_float3(1.f);
    prd.depth = 0;
    prd.hit = false;

    rtTrace(top_object, ray, prd);

    if(!prd.hit) {
      closest_intersection[launch_index] = -1;
      output_buffer[launch_index] = make_color(bg_color);
      return;
    }

    cur_world_loc = prd.world_loc;
    world_loc[launch_index] = cur_world_loc;

    acc_val += (make_float4(prd.result,0.f) - acc_val)/(frame+1);
    acc_val_occ += (make_float4(prd.occlusion,prd.t_hit) - acc_val_occ)/(frame+1);
    accum_buffer[launch_index] = acc_val;
    accum_buffer_occ[launch_index] = acc_val_occ;

    zmin = min(zmin, prd.shadow_intersection);
    closest_intersection[launch_index] = zmin;
  } else {
    cur_world_loc = world_loc[launch_index];
  }

  float4 blurred_occ = make_float4(0.0);
  //int2 pixel_radius = make_int2(5,5);
  //pixel_radius = make_int2(10,10);
  float sumWeight = 0.0;

  //i guess just blur here for now... inefficient, but gets the point across
  if (frame > brute_rpp || (frame > 0 && show_progressive)) {
    int numBlurred = 0;

    for(int i=-pixel_radius.x; i < pixel_radius.x; i++) {
      for(int j=-pixel_radius.y; j < pixel_radius.y; j++) {
        if(launch_index.x + i > 0 && launch_index.y + j > 0) {
          if(launch_index.x + i < output_buffer.size().x && launch_index.y + j < output_buffer.size().y) {
            float4 target_occ = accum_buffer_occ[make_uint2(launch_index.x+i, launch_index.y+j)];
            //float distance = target_occ.w - prd.t_hit;
            float3 loca = cur_world_loc;
            float3 locb = world_loc[make_uint2(launch_index.x+i, launch_index.y+j)];
            float3 diff = loca-locb;
            float distance = sqrt(diff.x*diff.x + diff.y*diff.y + diff.z*diff.z);
            if(distance < 0)
              distance = -distance;
            if (zmin > 0.0) {
              float weight = gaussFilter(distance,zmin);
              blurred_occ += weight * target_occ;
              sumWeight += weight;
              if (weight > 0)
                numBlurred += 1;
            }
          }
        }
      }
    }
    if(sumWeight > 0)
      blurred_occ /= sumWeight;
    if(err_vis && numBlurred < 2)
      blurred_occ = make_float4(1,0,0,1);
    /*
    if(err_vis)
      //blurred_occ = make_float4(closest_intersection[launch_index]/10.0);
      blurred_occ = make_float4(prd.shadow_intersection/10.0);
      */
    if(!blur_occ)
      blurred_occ = acc_val_occ;
  } else if (frame == 0){
    closest_intersection[launch_index] = 100000.0f;
    blurred_occ = acc_val_occ;
  } else {
    blurred_occ = acc_val_occ;
  }


  output_buffer[launch_index] = make_color( make_float3(acc_val)*make_float3(blurred_occ.x, blurred_occ.y, blurred_occ.z));


}


//
// Returns solid color for miss rays
//
RT_PROGRAM void miss()
{
  prd_radiance.result = bg_color;
  prd_radiance.shadow_intersection = 100000.0f;
}

//
// Terminates and fully attenuates ray after any hit
//
RT_PROGRAM void any_hit_shadow()

{
  // this material is opaque, so it fully attenuates all shadow rays
  prd_shadow.attenuation = make_float3(0);
  prd_shadow.distance = t_hit;

  rtTerminateRay();
}


//
// Phong surface shading with shadows 
//
rtDeclareVariable(float3,   Ka, , ); 
rtDeclareVariable(float3,   Ks, , ); 
rtDeclareVariable(float,    phong_exp, , );
rtDeclareVariable(float3,   Kd, , ); 
rtDeclareVariable(float3,   ambient_light_color, , );
rtBuffer<BoxLight>        lights;
rtDeclareVariable(rtObject, top_shadower, , );
rtDeclareVariable(float3, reflectivity, , );
rtDeclareVariable(float, importance_cutoff, , );
rtDeclareVariable(int, max_depth, , );

//asdf
rtBuffer<uint2, 2> shadow_rng_seeds;

RT_PROGRAM void closest_hit_radiance3()
{
  float3 world_geo_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
  float3 world_shade_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float3 ffnormal     = faceforward( world_shade_normal, -ray.direction, world_geo_normal );
  float3 color = Ka * ambient_light_color;

  float3 hit_point = ray.origin + t_hit * ray.direction;
  prd_radiance.t_hit = t_hit;
  prd_radiance.world_loc = hit_point;
  prd_radiance.hit = true;

  float3 occlusion = make_float3(0.0);
  for(int i = 0; i < lights.size(); ++i) {
    BoxLight light = lights[i];

    uint2 seed = shadow_rng_seeds[launch_index];
    //seed.x = rot_seed(seed.x, frame);
    //seed.y = rot_seed(seed.y, frame);
    float2 sample = make_float2( rnd(seed.x), rnd(seed.y) );

    shadow_rng_seeds[launch_index] = seed;

    for(int i=0; i<1; ++i) {

      //From point, choose a random direction to sample in
      float3 U, V, W;
      float3 sampleDir; 
      createONB( ffnormal, U, V, W); //(is ffnormal the correct one to be using here?)
      sampleUnitHemisphere( sample, U, V, W, sampleDir );
      prd_radiance.shadow_intersection = 100;

      float nDl = dot( ffnormal, sampleDir );

      if( nDl > 0.0f ){
        //compute color
        color += Kd;

        float3 H = normalize(sampleDir - ray.direction);
        float nDh = dot( ffnormal, H );
        if (nDh > 0)
          color += Ks * pow(nDh, phong_exp);



        //cast ray and check for shadow
        PerRayData_shadow shadow_prd;
        shadow_prd.attenuation = make_float3(1.0f);
        optix::Ray shadow_ray ( hit_point, sampleDir, shadow_ray_type, scene_epsilon );
        rtTrace(top_shadower, shadow_ray, shadow_prd);
        occlusion += shadow_prd.attenuation;
        prd_radiance.shadow_intersection = shadow_prd.distance;
      }
    }


  }
/*
  float importance = prd_radiance.importance * optix::luminance( reflectivity );

  if( importance > importance_cutoff && prd_radiance.depth < max_depth) {
    PerRayData_radiance refl_prd;
    refl_prd.importance = importance;
    refl_prd.depth = prd_radiance.depth+1;
    float3 R = reflect( ray.direction, ffnormal );
    optix::Ray refl_ray( hit_point, R, radiance_ray_type, scene_epsilon );
    rtTrace(top_object, refl_ray, refl_prd);
    //color += reflectivity * refl_prd.result;
  }*/



  prd_radiance.occlusion = occlusion;
  prd_radiance.result = color;
}


//
// Set pixel to solid color upon failure
//
RT_PROGRAM void exception()
{
  output_buffer[launch_index] = make_color( bad_color );
}
